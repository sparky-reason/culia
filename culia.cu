#include "hip/hip_runtime.h"
#include "culia.h"

#include ""
#include "surface_functions.h" 

#include <cstring>
#include <cstdint>

__global__ void test_kernel(hipSurfaceObject_t surface, int width, int height)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uint32_t data = 0xFF0000FF;
        surf2Dwrite(data, surface, 4*x, y);
    }
}

hipError_t render_julia_set(hipGraphicsResource_t cuda_renderbuffer, int width, int height)
{
    hipError_t cuda_err;

    // map buffer for writing from CUDA
    cuda_err = hipGraphicsMapResources(1, &cuda_renderbuffer);
    if (cuda_err != hipSuccess) return cuda_err;

    hipArray_t cuda_array;
    cuda_err = hipGraphicsSubResourceGetMappedArray(&cuda_array, cuda_renderbuffer, 0, 0);
    if (cuda_err != hipSuccess) return cuda_err;

    // create surface object
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuda_array;
    hipSurfaceObject_t cuda_surface;
    hipCreateSurfaceObject(&cuda_surface, &resDesc);

    // execute kernel
    dim3 dim_block(16, 16);
    dim3 dim_grid((width + dim_block.x - 1) / dim_block.x, (height + dim_block.y - 1) / dim_block.y);
    test_kernel<<<dim_grid, dim_block>>>(cuda_surface, width, height);

    // destroy surface object
    hipDestroySurfaceObject(cuda_surface);

    // unmap buffer
    hipGraphicsUnmapResources(1, &cuda_renderbuffer);

    return hipSuccess;
}